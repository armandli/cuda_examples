//matrix multiply using shared memory for optimization


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

#define BSZ 128
#define TSZ 16
#define SZ (BSZ * TSZ)
#define TT double

using namespace std;

template <typename T>
void random_matrix(T* m, size_t sz){
  srand(time(0));

  for (size_t i = 0; i < sz; ++i)
    m[i] = (TT)rand() / 100.F;
}

template <typename T>
struct Mtx {
public:
  T* data;
  size_t rows;
  size_t cols;
  bool is_cuda;

  Mtx(bool is_cuda, size_t rows, size_t cols):
    data(nullptr), rows(rows), cols(cols), is_cuda(is_cuda) {
    if (is_cuda) hipMalloc(&data, sizeof(T) * rows * cols);
    else         data = new T[rows * cols];
  }

  ~Mtx(){
    if (is_cuda) hipFree(data);
    else         delete[] data;
  }
};

template <typename T>
struct SubMtx {
  T* data;
  size_t rows;
  size_t cols;
  size_t stride;
};

template <typename T>
__device__ T get_elem(SubMtx<T>& a, size_t i, size_t j){
  return a.data[i * a.stride + j];
}

template <typename T>
__device__ void set_elem(SubMtx<T>& a, size_t i, size_t j, T val){
  a.data[i * a.stride + j] = val;
}

template <typename T>
__device__ SubMtx<T> sub_matrix_stride(Mtx<T>& m, size_t row_stride, size_t col_stride){
  SubMtx<T> ret;
  ret.data = &m.data[m.cols * TSZ * row_stride + TSZ * col_stride];
  ret.rows = ret.cols = TSZ;
  ret.stride = m.cols;
  return ret;
}

template <typename T>
__global__ void matrix_multiply_cuda_v2(Mtx<T>& c, Mtx<T>& a, Mtx<T>& b){
  size_t bx = blockIdx.x, by = blockIdx.y;
  SubMtx<T> csub = sub_matrix_stride(c, bx, by);

  T cval = 0.;

  size_t row = threadIdx.x, col = threadIdx.y;

  for (size_t i = 0; i < BSZ; ++i){
    SubMtx<T> asub = sub_matrix_stride(a, bx, i);
    SubMtx<T> bsub = sub_matrix_stride(b, i, by);

    __shared__ T amem[TSZ][TSZ];
    __shared__ T bmem[TSZ][TSZ];

    amem[row][col] = get_elem(asub, row, col);
    bmem[row][col] = get_elem(bsub, row, col);

    __syncthreads();

    for (size_t j = 0; j < TSZ; ++j)
      cval += amem[row][j] * bmem[j][col];

    __syncthreads();
  }

//  set_elem(csub, bx * blockDim.x + row, by * blockDim.y + col, cval);
  set_elem(csub, row, col, cval);
}

template <typename T>
clock_t matrix_multiply_v1(Mtx<T>& c, Mtx<T>& a, Mtx<T>& b){
  for (size_t i = 0; i < c.rows; ++i)
    for (size_t j = 0; j < c.cols; ++j){
      c.data[i * c.cols + j] = 0.;
      for (size_t k = 0; k < a.cols; ++k)
        c.data[i * c.cols + j] += a.data[i * a.cols + k] * b.data[k * b.cols + j];
    }

  return clock();
}

int main(){
  Mtx<TT> c(false, SZ, SZ), a(false, SZ, SZ), b(false, SZ, SZ), d(false, SZ, SZ);
  Mtx<TT> dc(true, SZ, SZ), da(true, SZ, SZ), db(true, SZ, SZ);

  random_matrix(a.data, SZ * SZ);
  random_matrix(b.data, SZ * SZ);

  clock_t timing_start = clock();

  hipMemcpy(da.data, a.data, sizeof(TT) * SZ * SZ, hipMemcpyHostToDevice);
  hipMemcpy(db.data, b.data, sizeof(TT) * SZ * SZ, hipMemcpyHostToDevice);

  dim3 dblock(BSZ, BSZ);
  dim3 dthread(BSZ, BSZ);
  matrix_multiply_cuda_v2<<<dblock, dthread>>>(dc, da, db);

  hipMemcpy(c.data, dc.data, sizeof(TT) * SZ * SZ, hipMemcpyDeviceToHost);

  cout << "CUDA time: " << (clock() - timing_start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

  timing_start = clock();

  clock_t timing_end = matrix_multiply_v1(d, a, b);

  cout << "CPU time: " << (timing_end - timing_start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

  bool match = true;
  for (size_t i = 0; i < SZ * SZ; ++i)
    if (c.data[i] - d.data[i] > 1e-5F){
      cout << "Values does not match" << endl;
      match = false;
      break;
    }

  if (match) cout << "All values match" << endl;
}
