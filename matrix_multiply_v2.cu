#include "hip/hip_runtime.h"
//matrix multiply using shared memory for optimization

#include <cstdlib>
#include <ctime>
#include <iostream>

#define BSZ 128
#define TSZ 16
#define SZ (BSZ * TSZ)
#define TT float

using namespace std;

template <typename T>
void random_matrix(T* m, size_t sz){
  srand(time(0));

  for (size_t i = 0; i < sz; ++i)
    m[i] = (TT)rand() / 100.F;
}

template <typename T>
struct Mtx {
public:
  T* data;
  size_t rows;
  size_t cols;
  size_t stride;
  int ownership_state;

  Mtx(): data(nullptr), rows(0), cols(0), stride(0), ownership_state(3) {}
  Mtx(bool is_cuda, size_t rows, size_t cols):
    data(nullptr), rows(rows), cols(cols), stride(cols), ownership_state(is_cuda ? 2 : 1) {
    if (is_cuda) hipMalloc(&data, sizeof(T) * rows * cols);
    else         data = new T[rows * cols];
  }

  ~Mtx(){
    switch (ownership_state){
      case 1: delete[] data; break;
      case 2: hipFree(data); break;
      case 3: break;
      default: assert(false);
    }
  }

//private:
//  Mtx(T* data, size_t rows, size_t cols, size_t stride):
//    data(data), rows(rows), cols(cols), stride(stride), ownership_state(3) {}
//public:
//  Mtx sub_matrix_stride(size_t row_stride, size_t col_stride){
//    return Mtx(&data[stride * TSZ * row_stride + TSZ * col_stride], TSZ, TSZ, stride);
//  }
};

template <typename T>
__device__ T get_elem(Mtx<T>& a, size_t i, size_t j){
  return a.data[i * a.cols + j];
}

template <typename T>
__device__ void set_elem(Mtx<T>& a, size_t i, size_t j, T val){
  a.data[i * a.cols + j] = val;
}

template <typename T>
__device__ Mtx<T> sub_matrix_stride(Mtx<T>& m, size_t row_stride, size_t col_stride){
  Mtx<T> ret;
  ret.data = &m.data[m.stride * TSZ * row_stride + TSZ * col_stride];
  ret.rows = ret.cols = TSZ;
  ret.stride = m.cols;
  return ret;
}

template <typename T>
__global__ void matrix_multiply_cuda_v2(Mtx<T>& c, Mtx<T>& a, Mtx<T>& b){
  size_t bx = blockIdx.x, by = blockIdx.y;
  Mtx<T> csub = sub_matrix_stride(c, bx, by);

  T cval = 0.;

  size_t row = threadIdx.x, col = threadIdx.y;

  for (size_t i = 0; i < BSZ; ++i){
    Mtx<T> asub = sub_matrix_stride(a, bx, i);
    Mtx<T> bsub = sub_matrix_stride(b, i, by);

    __shared__ T amem[TSZ][TSZ];
    __shared__ T bmem[TSZ][TSZ];

    amem[row][col] = get_elem(asub, row, col);
    bmem[row][col] = get_elem(bsub, row, col);

    __syncthreads();

    for (size_t j = 0; j < TSZ; ++j)
      cval += amem[row][j] * bmem[j][col];

    __syncthreads();
  }

  set_elem(csub, bx * blockDim.x + row, by * blockDim.y + col, cval);
}

template <typename T>
clock_t matrix_multiply_v1(Mtx<T>& c, Mtx<T>& a, Mtx<T>& b){
  for (size_t i = 0; i < c.rows; ++i)
    for (size_t j = 0; j < c.cols; ++j){
      c.data[i * c.cols + j] = 0.;
      for (size_t k = 0; k < a.cols; ++k)
        c.data[i * c.cols + j] += a.data[i * a.cols + k] * b.data[k * b.cols + j];
    }

  return clock();
}

int main(){
  Mtx<TT> c(false, SZ, SZ), a(false, SZ, SZ), b(false, SZ, SZ), d(false, SZ, SZ);
  Mtx<TT> dc(true, SZ, SZ), da(true, SZ, SZ), db(true, SZ, SZ);

  random_matrix(a.data, SZ * SZ);
  random_matrix(b.data, SZ * SZ);

  clock_t timing_start = clock();

  hipMemcpy(da.data, a.data, sizeof(TT) * SZ * SZ, hipMemcpyHostToDevice);
  hipMemcpy(db.data, b.data, sizeof(TT) * SZ * SZ, hipMemcpyHostToDevice);

  dim3 dblock(BSZ, BSZ);
  dim3 dthread(BSZ, BSZ);
  matrix_multiply_cuda_v2<<<dblock, dthread>>>(dc, da, db);

  hipMemcpy(c.data, dc.data, sizeof(TT) * SZ * SZ, hipMemcpyDeviceToHost);

  cout << "CUDA time: " << (clock() - timing_start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

  timing_start = clock();

  clock_t timing_end = matrix_multiply_v1(d, a, b);

  cout << "CPU time: " << (timing_end - timing_start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

  bool match = true;
  for (size_t i = 0; i < SZ * SZ; ++i)
    if (c.data[i] - d.data[i] > 1e-5F){
      cout << "Values does not match" << endl;
      match = false;
      break;
    }

  if (match) cout << "All values match" << endl;
}
